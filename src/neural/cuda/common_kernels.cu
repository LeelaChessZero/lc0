#include "hip/hip_runtime.h"
/*
  This file is part of Leela Chess Zero.
  Copyright (C) 2018-2019 The LCZero Authors

  Leela Chess is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  Leela Chess is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with Leela Chess.  If not, see <http://www.gnu.org/licenses/>.

  Additional permission under GNU GPL version 3 section 7

  If you modify this Program, or any covered work, by linking or
  combining it with NVIDIA Corporation's libraries from the NVIDIA CUDA
  Toolkit and the NVIDIA CUDA Deep Neural Network library (or a
  modified version of those libraries), containing parts covered by the
  terms of the respective license agreement, the licensors of this
  Program grant you additional permission to convey the resulting work.
*/

#include <algorithm>
#include <cassert>

#include "cuda_common.h"
#include "neural/shared/activation.h"
#include "neural/shared/attention_policy_map.h"
#include "winograd_helper.inc"

namespace lczero {
namespace cudnn_backend {
namespace {
constexpr int kInputPlanes = 112;
}  // namespace

/////////////////////////////////////////////////////////////////////////////
//          Simple CUDA kernels used by certain layers                     //
/////////////////////////////////////////////////////////////////////////////

template <typename T>
__global__ void addVectors_kernel(T* c, T* a, T* b, int size, int asize,
                                  int bsize, ActivationFunction activation) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < size) {
    float aVal = 0;
    float bVal = 0;
    if (a) aVal = (float)(a[i % asize]);
    if (b) bVal = (float)(b[i % bsize]);

    float cVal = aVal + bVal;

    cVal = activate(cVal, activation);

    c[i] = (T)cVal;
  }
}

// Adds two vectors (possibly of different sizes), also do optional relu
// activation.
template <typename T>
void addVectors(T* c, T* a, T* b, int size, int asize, int bsize,
                ActivationFunction activation, hipStream_t stream) {
  const int kBlockSize = 256;
  int blocks = DivUp(size, kBlockSize);

  addVectors_kernel<<<blocks, kBlockSize, 0, stream>>>(c, a, b, size, asize,
                                                       bsize, activation);
  ReportCUDAErrors(hipGetLastError());
}

template <typename T>
__global__ void addVectorsHNC_NHC_kernel(T* a, T* b, int N, int H, int C) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < N * H * C) {
    int orig_i = i;
    int c = i % C;
    i /= C;
    int n = i % N;
    i /= N;
    int h = i;
    float aVal = (float)a[orig_i];
    float bVal = (float)b[n * H * C + h * C + c];

    float cVal = aVal + bVal;

    a[orig_i] = (T)cVal;
  }
}

template <typename T>
void addVectorsHNC_NHC(T* a, T* b, int N, int H, int C, hipStream_t stream) {
  const int kBlockSize = 256;
  int blocks = DivUp(N * H * C, kBlockSize);
  addVectorsHNC_NHC_kernel<<<blocks, kBlockSize, 0, stream>>>(a, b, N, H, C);

  ReportCUDAErrors(hipGetLastError());
}

template <typename T, ActivationFunction act>
__global__ void addBiasBatched_kernel(T* output, const T* input, const T* bias,
                                      int N, int C) {
  int batch = blockIdx.y;
  int n = blockIdx.x * blockDim.y + threadIdx.y;
  if (n >= N) return;
  int c = threadIdx.x * 4;

  int biasIndex = batch * C + c;
  int tensorIndex = batch * N * C + n * C + c;

  float val[4];
  float b[4];

  // Load from memory
  const bool fp16 = std::is_same<half, T>::value;
  if (fp16) {
    half inp[4];
    copyAs<uint2>(&inp[0], &input[tensorIndex]);
#pragma unroll
    for (int i = 0; i < 4; i++) val[i] = (float)inp[i];

    copyAs<uint2>(&inp[0], &bias[biasIndex]);
#pragma unroll
    for (int i = 0; i < 4; i++) b[i] = (float)inp[i];
  } else {
    copyAs<uint4>(&val[0], &input[tensorIndex]);
    copyAs<uint4>(&b[0], &bias[biasIndex]);
  }

  // Perform bias add and activation
#pragma unroll
  for (int i = 0; i < 4; i++) {
    float x = val[i] + b[i];
    x = activate(x, act);
    val[i] = x;
  }

  // write to memory
  if (fp16) {
    half op[4];
#pragma unroll
    for (int i = 0; i < 4; i++) op[i] = (half)val[i];
    copyAs<uint2>(&output[tensorIndex], &op[0]);
  } else {
    copyAs<uint4>(&output[tensorIndex], &val[0]);
  }
}

// Input/output tensors are Batch * N * C
// bias tensor is N * C (i.e, different bias for each Batch dimension)
template <typename T>
void addBiasBatched(T* output, const T* input, const T* bias, int Batch, int N,
                    int C, ActivationFunction activation, hipStream_t stream) {
  // process 4 elements per thread to achieve close to peak memory bandwidth
  if (C % 4 != 0) throw Exception("unsupported filter size");
  if (C > 4096) throw Exception("unsupported filter size");

  dim3 blockDim, gridDim;
  blockDim.x = C / 4;
  blockDim.y = std::min(std::max(512 / blockDim.x, 1u), (unsigned int)N);
  blockDim.z = 1;
  gridDim.x = DivUp(N, blockDim.y);
  gridDim.y = Batch;
  gridDim.z = 1;

  switch (activation) {
    case ACTIVATION_NONE:
      addBiasBatched_kernel<T, ACTIVATION_NONE>
          <<<gridDim, blockDim, 0, stream>>>(output, input, bias, N, C);
      break;
    case ACTIVATION_SELU:
      addBiasBatched_kernel<T, ACTIVATION_SELU>
          <<<gridDim, blockDim, 0, stream>>>(output, input, bias, N, C);
      break;
    case ACTIVATION_MISH:
      addBiasBatched_kernel<T, ACTIVATION_MISH>
          <<<gridDim, blockDim, 0, stream>>>(output, input, bias, N, C);
      break;
    case ACTIVATION_RELU:
      addBiasBatched_kernel<T, ACTIVATION_RELU>
          <<<gridDim, blockDim, 0, stream>>>(output, input, bias, N, C);
      break;
    case ACTIVATION_SWISH:
      addBiasBatched_kernel<T, ACTIVATION_SWISH>
          <<<gridDim, blockDim, 0, stream>>>(output, input, bias, N, C);
      break;
    case ACTIVATION_RELU_2:  // square relu
      addBiasBatched_kernel<T, ACTIVATION_RELU_2>
          <<<gridDim, blockDim, 0, stream>>>(output, input, bias, N, C);
      break;
    default:
      throw Exception(
          "unsupported activation in addBiasBatched. Add in switch-case here");
  }

  ReportCUDAErrors(hipGetLastError());
}

template <typename T, ActivationFunction act>
__global__ void addBiasBatched_kernel(T* output, const T* input, const T* bias,
                                      int N, int C, int Nstride) {
  int batch = blockIdx.y;
  int n = blockIdx.x * blockDim.y + threadIdx.y;
  if (n >= N) return;
  int c = threadIdx.x * 4;

  int biasIndex = batch * C + c;
  int tensorIndex = batch * Nstride * C + n * C + c;

  float val[4];
  float b[4];

  // Load from memory
  const bool fp16 = std::is_same<half, T>::value;
  if (fp16) {
    half inp[4];
    copyAs<uint2>(&inp[0], &input[tensorIndex]);
#pragma unroll
    for (int i = 0; i < 4; i++) val[i] = (float)inp[i];

    copyAs<uint2>(&inp[0], &bias[biasIndex]);
#pragma unroll
    for (int i = 0; i < 4; i++) b[i] = (float)inp[i];
  } else {
    copyAs<uint4>(&val[0], &input[tensorIndex]);
    copyAs<uint4>(&b[0], &bias[biasIndex]);
  }

  // Perform bias add and activation
#pragma unroll
  for (int i = 0; i < 4; i++) {
    float x = val[i] + b[i];
    x = activate(x, act);
    val[i] = x;
  }

  // write to memory
  if (fp16) {
    half op[4];
#pragma unroll
    for (int i = 0; i < 4; i++) op[i] = (half)val[i];
    copyAs<uint2>(&output[tensorIndex], &op[0]);
  } else {
    copyAs<uint4>(&output[tensorIndex], &val[0]);
  }
}

// Input/output tensors are Batch * N * C
// bias tensor is N * C (i.e, different bias for each Batch dimension)
template <typename T>
void addBiasBatched(T* output, const T* input, const T* bias, int Batch, int N,
                    int C, int Nstride, ActivationFunction activation,
                    hipStream_t stream) {
  // process 4 elements per thread to achieve close to peak memory bandwidth
  if (C % 4 != 0) throw Exception("unsupported filter size");
  if (C > 4096) throw Exception("unsupported filter size");

  dim3 blockDim, gridDim;
  blockDim.x = C / 4;
  blockDim.y = std::min(std::max(512 / blockDim.x, 1u), (unsigned int)N);
  blockDim.z = 1;
  gridDim.x = DivUp(N, blockDim.y);
  gridDim.y = Batch;
  gridDim.z = 1;

  switch (activation) {
    case ACTIVATION_NONE:
      addBiasBatched_kernel<T, ACTIVATION_NONE>
          <<<gridDim, blockDim, 0, stream>>>(output, input, bias, N, C,
                                             Nstride);
      break;
    case ACTIVATION_SELU:
      addBiasBatched_kernel<T, ACTIVATION_SELU>
          <<<gridDim, blockDim, 0, stream>>>(output, input, bias, N, C,
                                             Nstride);
      break;
    case ACTIVATION_MISH:
      addBiasBatched_kernel<T, ACTIVATION_MISH>
          <<<gridDim, blockDim, 0, stream>>>(output, input, bias, N, C,
                                             Nstride);
      break;
    case ACTIVATION_RELU:
      addBiasBatched_kernel<T, ACTIVATION_RELU>
          <<<gridDim, blockDim, 0, stream>>>(output, input, bias, N, C,
                                             Nstride);
      break;
    case ACTIVATION_SWISH:
      addBiasBatched_kernel<T, ACTIVATION_SWISH>
          <<<gridDim, blockDim, 0, stream>>>(output, input, bias, N, C,
                                             Nstride);
      break;
    case ACTIVATION_RELU_2:  // square relu
      addBiasBatched_kernel<T, ACTIVATION_RELU_2>
          <<<gridDim, blockDim, 0, stream>>>(output, input, bias, N, C,
                                             Nstride);
      break;
    default:
      throw Exception(
          "unsupported activation in addBiasBatched. Add in switch-case here");
  }

  ReportCUDAErrors(hipGetLastError());
}

template <typename T>
__global__ void addBias_NCHW_kernel(T* c, T* a, T* b, int N, int C, int H,
                                    int W, ActivationFunction activation) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  int size = N * C * H * W;
  if (i < size) {
    float aVal = (float)a[i];

    // All this math can be optimized, but the kernel is memory bound anyway.
    int biasIndex = (i / (H * W)) % C;
    float bVal = (float)b[biasIndex];

    float cVal = aVal + bVal;

    cVal = activate(cVal, activation);

    c[i] = (T)cVal;
  }
}

// Add bias to convolution's output.
template <typename T>
void addBias_NCHW(T* c, T* a, T* b, int N, int C, int H, int W,
                  ActivationFunction activation, hipStream_t stream) {
  int size = N * C * H * W;
  const int kBlockSize = 256;
  int blocks = DivUp(size, kBlockSize);

  addBias_NCHW_kernel<<<blocks, kBlockSize, 0, stream>>>(c, a, b, N, C, H, W,
                                                         activation);
  ReportCUDAErrors(hipGetLastError());
}

template <typename dT, typename sT>
__device__ dT readNCHW(const sT* input_tensor, int n, int c, int h, int w,
                       int Nin, int Cin, int H, int W) {
  if (n >= Nin || c >= Cin) return 0;

  int index;
  index = n;
  index *= Cin;
  index += c;
  index *= H;
  index += h;
  index *= W;
  index += w;

  return (dT)(input_tensor[index]);
}

template <typename dT, typename sT>
__global__ void NCHWtoNHWC_kernel(dT* output_tensor, const sT* input_tensor,
                                  int Nin, int Cin, int Nout, int Cout, int H,
                                  int W) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid >= Nout * Cout * H * W) return;

  int index = tid;

  int c = (index % Cout);
  index /= Cout;
  int w = index % W;
  index /= W;
  int h = index % H;
  index /= H;
  int n = index;

  output_tensor[tid] =
      readNCHW<dT, sT>(input_tensor, n, c, h, w, Nin, Cin, H, W);
}

template <typename DstType, typename SrcType>
void convertNCHWtoNHWC(DstType* output_tensor, const SrcType* input_tensor,
                       int Nin, int Cin, int Nout, int Cout, int H, int W) {
  size_t numElements = Nout * Cout * H * W;
  const int blockSize = 256;
  int blocks = DivUp(numElements, blockSize);
  NCHWtoNHWC_kernel<<<blocks, blockSize>>>(output_tensor, input_tensor, Nin,
                                           Cin, Nout, Cout, H, W);
}

template <typename DstType, typename SrcType>
__global__ void copyTypeConverted_kernel(DstType* op, SrcType* ip, int N) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid >= N) return;

  DstType el = (DstType)ip[tid];
  op[tid] = el;
}

template <typename DstType, typename SrcType>
void copyTypeConverted(DstType* op, SrcType* ip, int N, hipStream_t stream) {
  const int kBlockSize = 256;
  int blocks = DivUp(N, kBlockSize);
  copyTypeConverted_kernel<<<blocks, kBlockSize, 0, stream>>>(op, ip, N);
}

template <typename T>
__global__ void batchNorm_kernel(T* output, const T* input, const T* skipInput,
                                 int N, int C, int H, int W, const float* means,
                                 const float* varMultipliers,
                                 ActivationFunction activation) {
  int index = threadIdx.x + blockDim.x * blockIdx.x;

  int wIndex = 0;
  if (sizeof(T) == sizeof(float))
    wIndex = (index / (H * W)) % C;  // NCHW for fp32.
  else
    wIndex = index % C;  // NHWC for fp16.

  float el = input[index];
  float mean = means[wIndex];
  float varMulti = varMultipliers[wIndex];

  el -= mean;
  el *= varMulti;

  if (skipInput) el += (float)skipInput[index];

  el = activate(el, activation);

  output[index] = (T)el;
}

// Every thread processes single element.
template <typename T>
void batchNorm(T* output, const T* input, const T* skipInput, int N, int C,
               int H, int W, float* means, float* var_multipliers,
               ActivationFunction activation) {
  const int total_elements = N * C * H * W;
  const int kBlockSize = 256;
  int blocks = DivUp(total_elements, kBlockSize);

  batchNorm_kernel<<<blocks, kBlockSize>>>(output, input, skipInput, N, C, H, W,
                                           means, var_multipliers, activation);

  ReportCUDAErrors(hipGetLastError());
}

__global__ void expandPlanes_kernel_Fp32_NCHW(float* output,
                                              const uint64_t* masks,
                                              const float* values, int n) {
  // Block size of 256, same mask/val for 64 consecutive threads.
  constexpr int kNumShmemElements = 256 / 64;

  __shared__ uint64_t shMasks[kNumShmemElements];
  __shared__ float shVals[kNumShmemElements];

  int index = threadIdx.x + blockDim.x * blockIdx.x;

  int planeIndex = index >> 6;

  if (planeIndex >= n) return;

  // Load inputs to shared memory.
  if (threadIdx.x < kNumShmemElements) {
    shMasks[threadIdx.x] = masks[planeIndex + threadIdx.x];
    shVals[threadIdx.x] = values[planeIndex + threadIdx.x];
  }
  __syncthreads();

  uint64_t mask = shMasks[threadIdx.x >> 6];

  int sqIndex = index & 0x3F;
  float op = 0;

  bool set = !!(mask & (1ull << sqIndex));
  if (set) {
    op = shVals[threadIdx.x >> 6];
  }
  output[index] = op;
}

void expandPlanes_Fp32_NCHW(float* output, const uint64_t* masks,
                            const float* values, int n, hipStream_t stream) {
  int threads = n * 8 * 8;  // Each thread writes a single element.
  const int blockSize = 256;
  int blocks = DivUp(threads, blockSize);
  expandPlanes_kernel_Fp32_NCHW<<<blocks, blockSize, 0, stream>>>(output, masks,
                                                                  values, n);
  ReportCUDAErrors(hipGetLastError());
}

// TODO: Can optimize using shared memory if this becomes a bottleneck.
__global__ void expandPlanes_kernel_Fp16_NHWC(half* output,
                                              const uint64_t* masks,
                                              const float* values, int n) {
  const int index = threadIdx.x + blockDim.x * blockIdx.x;
  if (index >= n * 8 * 8) return;

  const int planeIndex = index % kInputPlanes;
  const int boardIndex = index / (kInputPlanes * 8 * 8);
  const int sqIndex = (index / kInputPlanes) & 0x3F;

  uint64_t mask = masks[boardIndex * kInputPlanes + planeIndex];

  half op = 0;
  bool set = !!(mask & (1ull << sqIndex));
  if (set) {
    float val = values[boardIndex * kInputPlanes + planeIndex];
    op = (half)val;
  }
  output[index] = op;
}

void expandPlanes_Fp16_NHWC(half* output, const uint64_t* masks,
                            const float* values, int n, hipStream_t stream) {
  int threads = n * 8 * 8;  // Each thread writes a single element.
  const int kBlockSize = 256;
  int blocks = DivUp(threads, kBlockSize);
  expandPlanes_kernel_Fp16_NHWC<<<blocks, kBlockSize, 0, stream>>>(
      output, masks, values, n);
  ReportCUDAErrors(hipGetLastError());
}

__global__ void expandPlanes_kernel_Fp16_NCHW(half* output,
                                              const uint64_t* masks,
                                              const float* values, int n) {
  // block size of 256, same mask/val for 64 consecutive threads
  constexpr int kNumShmemElements = 256 / 64;

  __shared__ uint64_t shMasks[kNumShmemElements];
  __shared__ half shVals[kNumShmemElements];

  int index = threadIdx.x + blockDim.x * blockIdx.x;

  int planeIndex = index >> 6;

  if (planeIndex >= n) return;

  // load inputs to shared memory
  if (threadIdx.x < kNumShmemElements) {
    shMasks[threadIdx.x] = masks[planeIndex + threadIdx.x];
    shVals[threadIdx.x] = values[planeIndex + threadIdx.x];
  }
  __syncthreads();

  uint64_t mask = shMasks[threadIdx.x >> 6];

  int sqIndex = index & 0x3F;
  half op = 0;

  bool set = !!(mask & (1ull << sqIndex));
  if (set) {
    op = (half)shVals[threadIdx.x >> 6];
  }
  output[index] = op;
}

void expandPlanes_Fp16_NCHW(half* output, const uint64_t* masks,
                            const float* values, int n, hipStream_t stream) {
  int threads = n * 8 * 8;  // each thread writes a single element
  const int blockSize = 256;
  int blocks = DivUp(threads, blockSize);
  expandPlanes_kernel_Fp16_NCHW<<<blocks, blockSize, 0, stream>>>(output, masks,
                                                                  values, n);
  ReportCUDAErrors(hipGetLastError());
}

template <typename T>
__global__ void globalScale_kernel(T* output, const T* input,
                                   const T* scaleBias, const T* prevLayerBias,
                                   int inputSize, int C,
                                   ActivationFunction activation) {
  const int kPlaneSize = 64;

  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid > inputSize) return;

  int nc = tid / kPlaneSize;
  int n = nc / C;
  int c = nc % C;

  float val1 = input[tid];   // Output of residual block to be scaled.
  float val2 = output[tid];  // Skip connection to be added directly.

  if (prevLayerBias) {
    val1 += (float)(prevLayerBias[c]);
  }

  int startIdx = n * 2 * C;  // Scale and bias interleaved.

  float s = scaleBias[startIdx + c];
  s = 1.0f / (1.0f + exp(-s));  // Sigmoid on scale.

  float b = scaleBias[startIdx + c + C];

  float op = val1 * s + val2 + b;
  op = activate(op, activation);
  output[tid] = (T)op;
}

__global__ void globalScale_kernel_fp16_nhwc(half* output, const half* input,
                                             const half* scaleBias,
                                             const half* prevLayerBias,
                                             int inputSize, int C, int HWC,
                                             ActivationFunction activation) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid > inputSize) return;

  int c = tid % C;
  int n = tid / (HWC);

  float val1 = (float)input[tid];   // Output of residual block to be scaled.
  float val2 = (float)output[tid];  // Skip connection to be added directly.
  if (prevLayerBias) {
    val1 += (float)prevLayerBias[c];
  }

  int startIdx = n * 2 * C;  // Scale and bias interleaved.

  float s = scaleBias[startIdx + c];
  s = 1.0f / (1.0f + exp(-s));  // Sigmoid on scale.

  float b = scaleBias[startIdx + c + C];

  float op = val1 * s + val2 + b;
  op = activate(op, activation);

  output[tid] = (half)op;
}

// N blocks.
// C threads per block.
// 'HWC' input data processed by thread block.
// Each thread writes a single output.
__global__ void globalAvgPool_kernel_NHWC_fp16(half* output, const half* input,
                                               const half* prevLayerBias,
                                               int inputSize, int outputSize) {
  const int elementsPerThread = 64;  // 8x8 board.

  int blockStart = blockIdx.x * blockDim.x;

  float S = 0;

#pragma unroll
  for (int i = 0; i < elementsPerThread; i++) {
    int localIndex = i * blockDim.x + threadIdx.x;
    int inputIndex = blockStart * elementsPerThread + localIndex;
    if (inputIndex < inputSize) S += (float)(input[inputIndex]);
  }

  float avg = S / elementsPerThread;

  // Add bias from previous layer.
  if (prevLayerBias) avg += (float)(prevLayerBias[threadIdx.x]);

  int opIndex = blockStart + threadIdx.x;
  if (opIndex < outputSize) output[opIndex] = (half)avg;
}

// Each thread reads 2 inputs (8x8/32), and each warp writes a single output.
template <typename T>
__global__ void globalAvgPool_kernel(T* output, const T* input,
                                     const T* prevLayerBias, int inputSize,
                                     int outputSize, int C) {
  const int elementsPerWarp = 64;
  const int elementsPerThread = 2;

  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  int laneId = threadIdx.x & 0x1F;
  int laneStartIndex = (tid - laneId) * elementsPerThread;

  // Compute per-thread sum for elementsPerThread elements.
  float S = 0;

#pragma unroll
  for (int i = 0; i < elementsPerWarp; i += 32) {
    int index = laneStartIndex + laneId + i;
    if (index < inputSize) S += (float)(input[index]);
  }

// Compute warp wide sum (for entire plane - elementsPerWarp elements).
#pragma unroll
  for (int offset = 1; offset < 32; offset *= 2) {
    S += __shfl_down_sync(0xFFFFFFFF, S, offset);
  }

  float avg = S / elementsPerWarp;
  int opIndex = tid >> 5;

  // First thread in warp has the sum, write it in output.
  if (laneId == 0) {
    if (opIndex < outputSize) {
      if (prevLayerBias) avg += (float)prevLayerBias[opIndex % C];
      output[opIndex] = (T)avg;
    }
  }
}

template <typename T>
void globalAvgPool(int N, int C, T* output, const T* input,
                   const T* prevLayerBias, bool nhwc) {
  const int kPlaneSize = 64;
  if (nhwc) {
    assert((std::is_same<half, T>::value));
    // For NHWC fp16, simply launch N blocks, each with C threads.
    globalAvgPool_kernel_NHWC_fp16<<<N, C>>>((half*)output, (half*)input,
                                             (half*)prevLayerBias,
                                             N * C * kPlaneSize, N * C);
  } else {
    // For NCHW layout (used with fp32),
    // each warp processes a full plane (64 elements), and writes a single
    // average N*C warps are launched.

    const int kTotalWarps = N * C;
    const int kWarpsPerBlock = 8;
    const int kBlockSize = kWarpsPerBlock * 32;

    int blocks = DivUp(kTotalWarps, kWarpsPerBlock);
    globalAvgPool_kernel<<<blocks, kBlockSize>>>(output, input, prevLayerBias,
                                                 N * C * kPlaneSize, N * C, C);
  }
  ReportCUDAErrors(hipGetLastError());
}

template <typename T>
void globalScale(int N, int C, T* output, const T* input, const T* scaleBias,
                 const T* prevLayerBias, bool nhwc,
                 ActivationFunction activation) {
  // Each thread writes one output.
  const int kBlockSize = 256;
  const int kBlocks = DivUp(N * 8 * 8 * C, kBlockSize);

  if (nhwc) {
    assert((std::is_same<half, T>::value));
    globalScale_kernel_fp16_nhwc<<<kBlocks, kBlockSize>>>(
        (half*)output, (half*)input, (half*)scaleBias, (half*)prevLayerBias,
        N * C * 8 * 8, C, 8 * 8 * C, activation);
  } else {
    globalScale_kernel<<<kBlocks, kBlockSize>>>(
        output, input, scaleBias, prevLayerBias, N * C * 8 * 8, C, activation);
  }
  ReportCUDAErrors(hipGetLastError());
}

template <typename T>
__global__ void policyMap_kernel(T* output, const T* input,
                                 const short* indices, int N, int inputSize,
                                 int usedSize, int outputSize) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  int n = tid / usedSize;
  int i = tid % usedSize;

  if (n >= N) return;

  int j = indices[i];

  if (j >= 0) {
    output[n * outputSize + j] = input[n * inputSize + i];
  }
}

template <typename T>
void PolicyMap(int N, T* output, const T* input, const short* indices,
               int inputSize, int usedSize, int outputSize,
               hipStream_t stream) {
  // Each thread processes one input element
  // Only some of the threads (with valid mapping) write output
  const int kBlockSize = 256;
  const int kBlocks = DivUp(N * usedSize, kBlockSize);

  policyMap_kernel<T><<<kBlocks, kBlockSize, 0, stream>>>(
      (T*)output, (T*)input, (short*)indices, N, inputSize, usedSize,
      outputSize);
  ReportCUDAErrors(hipGetLastError());
}

template <typename T = float, bool use_se, ActivationFunction activation,
          bool use_bias, bool use_skip>
void OutputInputTransform(int N, int C, int se_K, T* output, const T* input,
                          const T* skip, const T* bias, const T* w1,
                          const T* b1, const T* w2, const T* b2,
                          hipStream_t stream) {
  // Each thread processes entire chess board
  if (use_se == false) {
    dim3 grid_dim(DivUp(C, kOpInpTransformBlockSize), N, 1);
    OutputTransform_relu_InputTransform_kernel<float, activation, use_bias,
                                               use_skip>
        <<<grid_dim, kOpInpTransformBlockSize, 0, stream>>>(N, C, output, input,
                                                            (float*)skip, bias);
  } else if (C > kMaxResBlockFusingChannels) {
    throw Exception(
        "res block fusing opt not supported for the given data type and no "
        "of filters\n");
  } else {
    OutputTransform_SE_relu_InputTransform_kernel<float, activation, use_bias,
                                                  use_skip>
        <<<N, C, 0, stream>>>(N, C, se_K, output, input, (float*)skip, bias, w1,
                              b1, w2, b2);
  }

  ReportCUDAErrors(hipGetLastError());
}

// softmax along C dimension which is assumed to be 64
// each thread processes two elements. Each warp computes a sum (over 64
// elements)
template <typename T>
__global__ void softmax_opt_64_kernel(T* output, const T* input,
                                      const T* input2, int N) {
  int index = blockDim.x * blockIdx.x + threadIdx.x;
  if (index >= N) return;

  float x[4];
  float ex[2];

  // Load from memory
  const bool fp16 = std::is_same<half, T>::value;
  if (fp16) {
    half inp[2];
    copyAs<int>(&inp[0], &input[index * 2]);
    x[0] = (float)inp[0];
    x[1] = (float)inp[1];
    if (input2 != nullptr) {
      copyAs<int>(&inp[0], &input2[index * 2]);
      x[2] = (float)inp[0];
      x[3] = (float)inp[1];
    }
  } else {
    copyAs<uint2>(&x[0], &input[index * 2]);
    if (input2 != nullptr) {
      copyAs<uint2>(&x[2], &input2[index * 2]);
    }
  }

  if (input2 != nullptr) {
    x[0] += x[2];
    x[1] += x[3];
  }
  float threadMax = max(x[0], x[1]);
  float maxval = warpMax(threadMax);
  maxval = __shfl_sync(0xFFFFFFFF, maxval, 0);

  ex[0] = exp(x[0] - maxval);
  ex[1] = exp(x[1] - maxval);

  float threadSum = ex[0] + ex[1];
  float Sum = warpReduce(threadSum);
  Sum = __shfl_sync(0xFFFFFFFF, Sum, 0);

  ex[0] = ex[0] / Sum;
  ex[1] = ex[1] / Sum;

  // Store to memory
  if (fp16) {
    half op[2];
    op[0] = (half)ex[0];
    op[1] = (half)ex[1];
    copyAs<int>(&output[index * 2], &op[0]);
  } else {
    copyAs<uint2>(&output[index * 2], &ex[0]);
  }
}

// N * C Tensors
// performs softmax along the C dimension
// Each thread processes one element
// Sums are computed in shared memory
// C threads per block, N blocks
template <typename T>
__global__ void softmax_kernel(T* output, const T* input, const T* input2) {
  int n = blockIdx.x;
  int c = threadIdx.x;
  int C = blockDim.x;
  int index = n * C + c;

  // softmax = tf.exp(logits) / tf.reduce_sum(tf.exp(logits), axis)

  float x = (float)input[index];
  if (input2 != nullptr) x += (float)input2[index];

  __shared__ float sum, maxval;
  if (c == 0) {
    sum = 0;
    maxval = x;
  }

  __syncthreads();

  // Get max across warp first, and then update across C dimension
  float warpmax = warpMax(x);
  if ((c & 0x1F) == 0) atomicMaxFloat(&maxval, warpmax);

  __syncthreads();

  float ex = exp(x - maxval);

  // compute warp wide sums first
  float val = warpReduce(ex);

  // update shared memory sum across C dimension
  if ((c & 0x1F) == 0) atomicAdd(&sum, val);

  __syncthreads();

  float op = ex / sum;

  output[index] = (T)op;
}

template <typename T>
void Softmax(int N, int C, T* output, const T* input, const T* input2,
             hipStream_t stream) {
  if (C == 64) {
    int size = N * 32;  // Total no of threads needed
    const int kBlockSize = 256;
    int blocks = DivUp(size, kBlockSize);
    softmax_opt_64_kernel<T>
        <<<blocks, kBlockSize, 0, stream>>>(output, input, input2, size);
  } else {
    softmax_kernel<T><<<N, C, 0, stream>>>(output, input, input2);
  }

  ReportCUDAErrors(hipGetLastError());
}

__device__ __forceinline__ float shared_sum_for_layer_norm(float x) {
  // compute warp-wide sum
  float s = warpReduce(x);

  // warp-wide sums
  // Max product of the two dimension for the below array is 16 (512/32), but
  // we make each dimension 16 for simplicity. if shared memory capacity is the
  // bottleneck (it's not), we can convert these to single dim array and
  // dynamically index
  __shared__ float sum[16][16];

  // compute sum across C dimension using the warp wide partial sums
  if (threadIdx.x == 0) sum[threadIdx.z][threadIdx.y] = s;
  __syncthreads();

  if (threadIdx.x == 0 && threadIdx.y == 0) {
    float cSum = 0;
    for (int j = 0; j < blockDim.y; j++) cSum += sum[threadIdx.z][j];
    sum[threadIdx.z][0] = cSum;
  }
  __syncthreads();

  // s now contains the sum across C dimension
  return sum[threadIdx.z][0];
}

// Each thread processes 4 elements
// 1. Perform Bias add, and skip add
// 2. Perform layer norm (normalize across C dimension)
template <typename T>
__global__ void layer_norm_kernel(int N, int C, T* output, const T* input,
                                  const T* bias, const T* skip, const T* gammas,
                                  const T* betas, float ep, float alpha,
                                  ActivationFunction act) {
  int n = blockIdx.x * blockDim.z + threadIdx.z;
  if (n >= N) return;
  int c = (threadIdx.y * 32 + threadIdx.x) * 16;
  bool oobThread = c >= C;

  int biasIndex = c;
  int tensorIndex = n * C + c;

  float val[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
  float oth[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

  const bool fp16 = std::is_same<half, T>::value;
  if (!oobThread) {
    // Load from memory (16 elements a time)
    if (fp16) {
      half inp[8];
      copyAs<uint4>(&inp[0], &input[tensorIndex]);
      for (int i = 0; i < 8; i++) val[i] = (float)inp[i];
      copyAs<uint4>(&inp[0], &input[tensorIndex + 8]);
      for (int i = 0; i < 8; i++) val[i + 8] = (float)inp[i];
      copyAs<uint4>(&inp[0], &bias[biasIndex]);
      for (int i = 0; i < 8; i++) oth[i] = (float)inp[i];
      copyAs<uint4>(&inp[0], &bias[biasIndex + 8]);
      for (int i = 0; i < 8; i++) oth[i + 8] = (float)inp[i];
      for (int i = 0; i < 16; i++) val[i] += oth[i];
    } else {
      copyAs<uint4>(&val[0], &input[tensorIndex]);
      copyAs<uint4>(&val[4], &input[tensorIndex + 4]);
      copyAs<uint4>(&val[8], &input[tensorIndex + 8]);
      copyAs<uint4>(&val[12], &input[tensorIndex + 12]);
      copyAs<uint4>(&oth[0], &bias[biasIndex]);
      copyAs<uint4>(&oth[4], &bias[biasIndex + 4]);
      copyAs<uint4>(&oth[8], &bias[biasIndex + 8]);
      copyAs<uint4>(&oth[12], &bias[biasIndex + 12]);
      for (int i = 0; i < 16; i++) val[i] += oth[i];
    }
  }

  if (!oobThread) {
    if (skip != nullptr) {
      // Load from memory (16 elements a time)
      if (fp16) {
        half inp[8];
        copyAs<uint4>(&inp[0], &skip[tensorIndex]);
        for (int i = 0; i < 8; i++) oth[i] = (float)inp[i];
        copyAs<uint4>(&inp[0], &skip[tensorIndex + 8]);
        for (int i = 0; i < 8; i++) oth[i + 8] = (float)inp[i];
      } else {
        copyAs<uint4>(&oth[0], &skip[tensorIndex]);
        copyAs<uint4>(&oth[4], &skip[tensorIndex + 4]);
        copyAs<uint4>(&oth[8], &skip[tensorIndex + 8]);
        copyAs<uint4>(&oth[12], &skip[tensorIndex + 12]);
      }
    }
  }

  // 1. Compute mean
  float s = 0;
  if (!oobThread)
    if (skip != nullptr) {
      for (int i = 0; i < 16; i++) {
        val[i] = activate(val[i], act) * alpha + oth[i];
        s += val[i];
      }
    } else {
      for (int i = 0; i < 16; i++) {
        val[i] = activate(val[i], act) * alpha;
        s += val[i];
      }
    }

  s = shared_sum_for_layer_norm(s);
  float mean = s / C;

  // 2. Compute varience
  s = 0;
  if (!oobThread)
    for (int i = 0; i < 16; i++) {
      float d = val[i] - mean;
      float d_sq = d * d;
      s += d_sq;
    }
  s = shared_sum_for_layer_norm(s);
  float var = s / C;

  if (!oobThread) {
    // Load from memory (16 elements a time)
    if (fp16) {
      half inp[8];
      copyAs<uint4>(&inp[0], &gammas[biasIndex]);
      for (int i = 0; i < 8; i++) oth[i] = (float)inp[i];
      copyAs<uint4>(&inp[0], &gammas[biasIndex + 8]);
      for (int i = 0; i < 8; i++) oth[i + 8] = (float)inp[i];
    } else {
      copyAs<uint4>(&oth[0], &gammas[biasIndex]);
      copyAs<uint4>(&oth[4], &gammas[biasIndex + 4]);
      copyAs<uint4>(&oth[8], &gammas[biasIndex + 8]);
      copyAs<uint4>(&oth[12], &gammas[biasIndex + 12]);
    }
  }

  // 3. Normalize
  for (int i = 0; i < 16; i++) {
    float d = val[i] - mean;
    float norm = d / sqrt(var + ep);
    float op = norm * oth[i];
    val[i] = op;
  }

  if (!oobThread) {
    // Load from memory (16 elements a time)
    if (fp16) {
      half inp[8];
      copyAs<uint4>(&inp[0], &betas[biasIndex]);
      for (int i = 0; i < 8; i++) oth[i] = (float)inp[i];
      copyAs<uint4>(&inp[0], &betas[biasIndex + 8]);
      for (int i = 0; i < 8; i++) oth[i + 8] = (float)inp[i];
    } else {
      copyAs<uint4>(&oth[0], &betas[biasIndex]);
      copyAs<uint4>(&oth[4], &betas[biasIndex + 4]);
      copyAs<uint4>(&oth[8], &betas[biasIndex + 8]);
      copyAs<uint4>(&oth[12], &betas[biasIndex + 12]);
    }
  }

  for (int i = 0; i < 16; i++) {
    val[i] += oth[i];
  }

  if (!oobThread) {
    // Write to memory
    if (fp16) {
      half op[8];
      for (int i = 0; i < 8; i++) op[i] = (half)val[i];
      copyAs<uint4>(&output[tensorIndex], &op[0]);
      for (int i = 0; i < 8; i++) op[i] = (half)val[i + 8];
      copyAs<uint4>(&output[tensorIndex + 8], &op[0]);
    } else {
      copyAs<uint4>(&output[tensorIndex], &val[0]);
      copyAs<uint4>(&output[tensorIndex + 4], &val[4]);
      copyAs<uint4>(&output[tensorIndex + 8], &val[8]);
      copyAs<uint4>(&output[tensorIndex + 12], &val[12]);
    }
  }
}

// add (optional) skip connection to input, and then perform Layer normalization
// normalization is done across C dimension (i.e, sums and std deviations taken
// over elements in C dim)
template <typename T>
void LayerNorm(int N, int C, T* output, const T* input, const T* bias,
               const T* skip, const T* gammas, const T* betas, float ep,
               float alpha, ActivationFunction act, hipStream_t stream) {
  // process 4 elements per thread to achieve close to peak memory bandwidth
  if (C % 16 != 0) throw Exception("unsupported filter size");
  if (C > 16384) throw Exception("unsupported filter size");

  dim3 blockDim, gridDim;
  blockDim.x = 32;
  blockDim.y = DivUp(C / 16, 32);
  blockDim.z =
      std::min(std::max(512 / (blockDim.x * blockDim.y), 1u), (unsigned int)N);
  gridDim.x = DivUp(N, blockDim.z);
  gridDim.y = 1;
  gridDim.z = 1;

  layer_norm_kernel<T><<<gridDim, blockDim, 0, stream>>>(
      N, C, output, input, bias, skip, gammas, betas, ep, alpha, act);

  ReportCUDAErrors(hipGetLastError());
}

// Compute promotion logits in a single kernel
// keys matrix is of N * 64 * C (but we use only last 8 from the 'rows'
// dimension, so N * 8 * C)
// ppo matrix is 4 * C (weights for dense layer / matrix multiplication)
// policy_attn_logits matrix is N * 64 * 64, but we use only 8x8 part of it
// from each batch dimension (so, N * 8 * 8)
// output matrix (promotion logits) is of N * 8 * 24 size
template <typename T>
__global__ void promotion_logits_kernel(int C, T* output, const T* keys,
                                        const T* ppo,
                                        const T* policy_attn_logits) {
  constexpr int output_stride = 64 * 64 + 8 * 24;
  int n = blockIdx.x;   // [0..N)
  int y = threadIdx.y;  // [0..8)
  int x = threadIdx.x;  // [0..24)     // Can split into 8 * 3

  int threadInGroup = threadIdx.y * 24 + threadIdx.x;

  // phase 1 : compute promotion_offsets by multiplying keys and ppo matrices
  const T* keys_start =
      keys + n * 64 * C + C * 56;  // we are interested only in last 8 out of 64
                                   // 'rows' of keys matrix
  __shared__ float promotion_offsets[4][8];

  // only 32 threads out of 192 in the group are active in this phase, and each
  // thread computes one element of the promotion_offsets matrix
  // TODO: opt idea1, can use more threads to reduce the length of the loop for
  // the matrix multiply (do parallel reduction of partial sums later)
  //       opt idea2, the below loop for matrix mul has very poor memory access
  //       pattern, can do the loop over 32, and do parallel reductions
  if (threadInGroup < 32) {
    int x = threadInGroup % 4;
    int y = threadInGroup / 4;

    float S = 0;
    for (int i = 0; i < C;
         i++) {  // TODO: modify to loop over 32 instead of C (doing parallel
                 // reductions for the 32 sums)
      float a = (float)keys_start[y * C + i];
      float b =
          (float)ppo[x * C + i];  // weight matrix is transposed (col major)
      S += a * b;
    }

    // write the product (promotion_offsets) in shared memory
    promotion_offsets[x][y] = S;
  }

  __syncthreads();

  // phase 2: add the last "row" to the other 3
  // #knight offset is added to the other three
  // promotion_offsets = promotion_offsets[:, :3, :] + promotion_offsets[:, 3:4,
  // :]
  // Only 24 threads in the group are active in this phase
  if (threadInGroup < 32) {
    int x = threadInGroup % 4;
    int y = threadInGroup / 4;
    if (x < 3) {
      promotion_offsets[x][y] += promotion_offsets[3][y];
    }
  }

  __syncthreads();

  // phase 3: add 8x8 chunk of policy_attn_logits matrix to promotion offsets
  //          the output is 3x8x8 (written as 8 * 24)
  // All threads are active in this phase and they compute one element each
  int w = x / 3;
  int c = x % 3;

  // n_promo_logits = matmul_qk[:, -16:-8, -8:]  # default traversals from rank
  // 7 to rank 8
  float n_promo_logit =
      (float)policy_attn_logits[n * output_stride + (48 + y) * 64 + (56 + w)];
  float promo_offset = promotion_offsets[c][w];

  float op = n_promo_logit + promo_offset;

  output[n * output_stride + threadInGroup] = (T)op;
}

template <typename T>
void ComputePromotionLogits(int N, int C, T* output, const T* keys,
                            const T* ppo, const T* policy_attn_logits,
                            hipStream_t stream) {
  // N blocks
  // 8 * 24 threads
  // Each thread computes a single output element
  dim3 blockDim(24, 8, 1);
  promotion_logits_kernel<T>
      <<<N, blockDim, 0, stream>>>(C, output, keys, ppo, policy_attn_logits);
}

template <typename T>
__global__ void preprocess_for_attention_body_kernel(
    T* output, const T* input, const T* encoding, int input_size,
    int encoding_size, bool is_pe_dense_embedding) {
  int n = blockIdx.x;
  int hw = blockIdx.y;
  int c = threadIdx.x;

  T op;
  if (c >= input_size) {
    // concatenate from position encoding array
    if (is_pe_dense_embedding) {
      op = (T)(encoding[n * 64 * encoding_size + hw * encoding_size + (c - input_size)]);
    } else {
      op = (T)(encoding[64 * hw + (c - input_size)]);
    }
  } else {
    op = input[n * input_size * 64 + c * 64 + hw];  // nchw
  }

  int outputC = input_size + encoding_size;

  // convert to nhwc
  output[n * 64 * outputC + hw * outputC + c] = op;
}

template <typename T>
void inputPreprocessForAttentionBody(T* output, const T* input,
                                     const T* encoding, int N, int input_size,
                                     int encoding_size,
                                     bool is_pe_dense_embedding,
                                     hipStream_t stream) {
  // N * 64 blocks
  // (kInputPlanes + kNumPosEncodingChannels) threads
  // Each thread computes a single output element
  dim3 gridSize = dim3(N, 64);
  int blockSize = input_size + encoding_size;
  preprocess_for_attention_body_kernel<T><<<gridSize, blockSize, 0, stream>>>(
      output, input, encoding, input_size, encoding_size,
      is_pe_dense_embedding);
}

template <typename T>
__global__ void input_gating_kernel(T* output, const T* input, const T* mult,
                                    const T* add, int HW, int C) {
  int n_offset = blockIdx.z * HW * C;
  int idx = threadIdx.y * C + blockIdx.x * blockDim.x +
            threadIdx.x;  // index in input
  int idxT = (blockIdx.x * blockDim.x + threadIdx.x) * HW +
             threadIdx.y;  // index in transposed weights arrays mult and add.

  if (idx < HW * C) {
    // Combine multiply gating, add gating and weights transpose.
    float op =
        (float)input[n_offset + idx] * (float)mult[idxT] + (float)add[idxT];
    output[n_offset + idx] = (T)op;
  }
}

template <typename T>
void applyInputGating(T* output, const T* input, const T* mult, const T* add,
                      int N, int HW, int C, hipStream_t stream) {
  // Multiple blocks to fit into each input area / volume
  // Block x position indicates horizontal section of area
  // Block y position indicates batch
  // Each thread computes a single output element
  dim3 blockSize, gridSize;
  blockSize.x = DivUp(1024, HW);
  blockSize.y = HW;
  blockSize.z = 1;
  gridSize.x = DivUp(C, blockSize.x);
  gridSize.y = 1;
  gridSize.z = N;
  input_gating_kernel<T>
      <<<gridSize, blockSize, 0, stream>>>(output, input, mult, add, HW, C);

  ReportCUDAErrors(hipGetLastError());
}

// Template instantiation.
template void copyTypeConverted<half, float>(half* op, float* ip, int N,
                                             hipStream_t stream);
template void copyTypeConverted<float, half>(float* op, half* ip, int N,
                                             hipStream_t stream);
template void copyTypeConverted<float, float>(float* op, float* ip, int N,
                                              hipStream_t stream);
template void copyTypeConverted<half, half>(half* op, half* ip, int N,
                                            hipStream_t stream);

template void batchNorm<float>(float* output, const float* input,
                               const float* skipInput, int N, int C, int H,
                               int W, float* means, float* var_multipliers,
                               ActivationFunction activation);
template void batchNorm<half>(half* output, const half* input,
                              const half* skipInput, int N, int C, int H, int W,
                              float* means, float* var_multipliers,
                              ActivationFunction activation);

template void addVectors<float>(float* c, float* a, float* b, int size,
                                int asize, int bsize, ActivationFunction act,
                                hipStream_t stream);
template void addVectors<half>(half* c, half* a, half* b, int size, int asize,
                               int bsize, ActivationFunction act,
                               hipStream_t stream);

template void addVectorsHNC_NHC<float>(float* a, float* b, int N, int H, int C,
                                       hipStream_t stream);
template void addVectorsHNC_NHC<half>(half* a, half* b, int N, int H, int C,
                                      hipStream_t stream);

template void addBiasBatched<float>(float* output, const float* input,
                                    const float* bias, int Batch, int N, int C,
                                    ActivationFunction activation,
                                    hipStream_t stream);
template void addBiasBatched<half>(half* output, const half* input,
                                   const half* bias, int Batch, int N, int C,
                                   ActivationFunction activation,
                                   hipStream_t stream);

template void addBiasBatched<float>(float* output, const float* input,
                                    const float* bias, int Batch, int N, int C,
                                    int Nstride, ActivationFunction activation,
                                    hipStream_t stream);
template void addBiasBatched<half>(half* output, const half* input,
                                   const half* bias, int Batch, int N, int C,
                                   int Nstride, ActivationFunction activation,
                                   hipStream_t stream);

template void addBias_NCHW<float>(float* c, float* a, float* b, int N, int C,
                                  int H, int W, ActivationFunction activation,
                                  hipStream_t stream);

template void addBias_NCHW<half>(half* c, half* a, half* b, int N, int C, int H,
                                 int W, ActivationFunction activation,
                                 hipStream_t stream);

template void globalAvgPool<float>(int N, int C, float* output,
                                   const float* input,
                                   const float* prevLayerBias, bool nhwc);
template void globalAvgPool<half>(int N, int C, half* output, const half* input,
                                  const half* prevLayerBias, bool nhwc);

template void globalScale<float>(int N, int C, float* output,
                                 const float* input, const float* scaleBias,
                                 const float* prevLayerBias, bool nhwc,
                                 ActivationFunction activation);
template void globalScale<half>(int N, int C, half* output, const half* input,
                                const half* scaleBias,
                                const half* prevLayerBias, bool nhwc,
                                ActivationFunction activation);

template void PolicyMap<float>(int N, float* output, const float* input,
                               const short* indices, int inputSize,
                               int usedSize, int outputSize,
                               hipStream_t stream);

template void PolicyMap<half>(int N, half* output, const half* input,
                              const short* indices, int inputSize, int usedSize,
                              int outputSize, hipStream_t stream);

template void FilterTransform<float>(int N, int C, float* transformedFilter,
                                     const float* filter);

template void InputTransform<float, true>(int N, int C,
                                          float* transformed_input,
                                          const float* input,
                                          hipStream_t stream);

template void InputTransform<float, false>(int N, int C,
                                           float* transformed_input,
                                           const float* input,
                                           hipStream_t stream);

template void OutputTransform<float, true, ACTIVATION_RELU, true, true, false,
                              false>(int N, int C, int se_K, float* output,
                                     const float* input, const float* skip,
                                     const float* bias, const float* w1,
                                     const float* b1, const float* w2,
                                     const float* b2, hipStream_t stream);

template void
OutputTransform<float, false, ACTIVATION_RELU, true, true, false, false>(

    int N, int C, int se_K, float* output, const float* input,
    const float* skip, const float* bias, const float* w1, const float* b1,
    const float* w2, const float* b2, hipStream_t stream);

template void OutputTransform<float, true, ACTIVATION_RELU, true, true, true,
                              false>(int N, int C, int se_K, float* output,
                                     const float* input, const float* skip,
                                     const float* bias, const float* w1,
                                     const float* b1, const float* w2,
                                     const float* b2, hipStream_t stream);

template void OutputTransform<float, false, ACTIVATION_RELU, true, true, true,
                              false>(int N, int C, int se_K, float* output,
                                     const float* input, const float* skip,
                                     const float* bias, const float* w1,
                                     const float* b1, const float* w2,
                                     const float* b2, hipStream_t stream);

template void OutputTransform<float, false, ACTIVATION_RELU, true, false, false,
                              false>(int N, int C, int se_K, float* output,
                                     const float* input, const float* skip,
                                     const float* bias, const float* w1,
                                     const float* b1, const float* w2,
                                     const float* b2, hipStream_t stream);

template void OutputTransform<float, false, ACTIVATION_RELU, true, false, false,
                              true>(int N, int C, int se_K, float* output,
                                    const float* input, const float* skip,
                                    const float* bias, const float* w1,
                                    const float* b1, const float* w2,
                                    const float* b2, hipStream_t stream);

template void OutputTransform<float, true, ACTIVATION_RELU, true, true, true,
                              true>(int N, int C, int se_K, float* output,
                                    const float* input, const float* skip,
                                    const float* bias, const float* w1,
                                    const float* b1, const float* w2,
                                    const float* b2, hipStream_t stream);

template void OutputTransform<float, true, ACTIVATION_MISH, true, true, false,
                              false>(int N, int C, int se_K, float* output,
                                     const float* input, const float* skip,
                                     const float* bias, const float* w1,
                                     const float* b1, const float* w2,
                                     const float* b2, hipStream_t stream);

template void OutputTransform<float, false, ACTIVATION_MISH, true, true, false,
                              false>(int N, int C, int se_K, float* output,
                                     const float* input, const float* skip,
                                     const float* bias, const float* w1,
                                     const float* b1, const float* w2,
                                     const float* b2, hipStream_t stream);

template void OutputTransform<float, true, ACTIVATION_MISH, true, true, true,
                              false>(int N, int C, int se_K, float* output,
                                     const float* input, const float* skip,
                                     const float* bias, const float* w1,
                                     const float* b1, const float* w2,
                                     const float* b2, hipStream_t stream);

template void OutputTransform<float, false, ACTIVATION_MISH, true, true, true,
                              false>(int N, int C, int se_K, float* output,
                                     const float* input, const float* skip,
                                     const float* bias, const float* w1,
                                     const float* b1, const float* w2,
                                     const float* b2, hipStream_t stream);

template void OutputTransform<float, false, ACTIVATION_MISH, true, false, false,
                              false>(int N, int C, int se_K, float* output,
                                     const float* input, const float* skip,
                                     const float* bias, const float* w1,
                                     const float* b1, const float* w2,
                                     const float* b2, hipStream_t stream);

template void OutputTransform<float, false, ACTIVATION_MISH, true, false, false,
                              true>(int N, int C, int se_K, float* output,
                                    const float* input, const float* skip,
                                    const float* bias, const float* w1,
                                    const float* b1, const float* w2,
                                    const float* b2, hipStream_t stream);

template void OutputTransform<float, true, ACTIVATION_MISH, true, true, true,
                              true>(int N, int C, int se_K, float* output,
                                    const float* input, const float* skip,
                                    const float* bias, const float* w1,
                                    const float* b1, const float* w2,
                                    const float* b2, hipStream_t stream);

template void OutputTransform<float, false, ACTIVATION_NONE, true, false, false,
                              false>(int N, int C, int se_K, float* output,
                                     const float* input, const float* skip,
                                     const float* bias, const float* w1,
                                     const float* b1, const float* w2,
                                     const float* b2, hipStream_t stream);

template void OutputInputTransform<float, true, ACTIVATION_RELU, true, true>(
    int N, int C, int se_K, float* output, const float* input,
    const float* skip, const float* bias, const float* w1, const float* b1,
    const float* w2, const float* b2, hipStream_t stream);

template void OutputInputTransform<float, false, ACTIVATION_RELU, true, true>(
    int N, int C, int se_K, float* output, const float* input,
    const float* skip, const float* bias, const float* w1, const float* b1,
    const float* w2, const float* b2, hipStream_t stream);

template void OutputInputTransform<float, false, ACTIVATION_RELU, true, false>(
    int N, int C, int se_K, float* output, const float* input,
    const float* skip, const float* bias, const float* w1, const float* b1,
    const float* w2, const float* b2, hipStream_t stream);

template void OutputInputTransform<float, true, ACTIVATION_MISH, true, true>(
    int N, int C, int se_K, float* output, const float* input,
    const float* skip, const float* bias, const float* w1, const float* b1,
    const float* w2, const float* b2, hipStream_t stream);

template void OutputInputTransform<float, false, ACTIVATION_MISH, true, true>(
    int N, int C, int se_K, float* output, const float* input,
    const float* skip, const float* bias, const float* w1, const float* b1,
    const float* w2, const float* b2, hipStream_t stream);

template void OutputInputTransform<float, false, ACTIVATION_MISH, true, false>(
    int N, int C, int se_K, float* output, const float* input,
    const float* skip, const float* bias, const float* w1, const float* b1,
    const float* w2, const float* b2, hipStream_t stream);

template void Softmax<half>(int N, int C, half* output, const half* input,
                            const half* input2, hipStream_t stream);
template void Softmax<float>(int N, int C, float* output, const float* input,
                             const float* input2, hipStream_t stream);

template void LayerNorm<half>(int N, int C, half* output, const half* input,
                              const half* bias, const half* skip,
                              const half* gammas, const half* betas, float ep,
                              float alpha, ActivationFunction act,
                              hipStream_t stream);
template void LayerNorm<float>(int N, int C, float* output, const float* input,
                               const float* bias, const float* skip,
                               const float* gammas, const float* betas,
                               float ep, float alpha, ActivationFunction act,
                               hipStream_t stream);

template void ComputePromotionLogits<half>(int N, int C, half* output,
                                           const half* keys, const half* ppo,
                                           const half* policy_attn_logits,
                                           hipStream_t stream);
template void ComputePromotionLogits<float>(int N, int C, float* output,
                                            const float* keys, const float* ppo,
                                            const float* policy_attn_logits,
                                            hipStream_t stream);

template void convertNCHWtoNHWC<half, float>(half* output_tensor,
                                             const float* input_tensor, int Nin,
                                             int Cin, int Nout, int Cout, int H,
                                             int W);
template void convertNCHWtoNHWC<float, float>(float* output_tensor,
                                              const float* input_tensor,
                                              int Nin, int Cin, int Nout,
                                              int Cout, int H, int W);
template void convertNCHWtoNHWC<half, half>(half* output_tensor,
                                            const half* input_tensor, int Nin,
                                            int Cin, int Nout, int Cout, int H,
                                            int W);

template void inputPreprocessForAttentionBody<half>(
    half* output, const half* input, const half* encoding, int N,
    int input_size, int encoding_size, bool is_pe_dense_embedding,
    hipStream_t stream);

template void inputPreprocessForAttentionBody<float>(
    float* output, const float* input, const float* encoding, int N,
    int input_size, int encoding_size, bool is_pe_dense_embedding,
    hipStream_t stream);

template void applyInputGating<half>(half* output, const half* input,
                                     const half* mult, const half* add, int N,
                                     int C, int output_size,
                                     hipStream_t stream);

template void applyInputGating<float>(float* output, const float* input,
                                      const float* mult, const float* add,
                                      int N, int C, int output_size,
                                      hipStream_t stream);
}  // namespace cudnn_backend
}  // namespace lczero
