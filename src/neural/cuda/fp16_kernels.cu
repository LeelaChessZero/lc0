#include "hip/hip_runtime.h"
/*
  This file is part of Leela Chess Zero.
  Copyright (C) 2018 The LCZero Authors

  Leela Chess is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  Leela Chess is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with Leela Chess.  If not, see <http://www.gnu.org/licenses/>.

  Additional permission under GNU GPL version 3 section 7

  If you modify this Program, or any covered work, by linking or
  combining it with NVIDIA Corporation's libraries from the NVIDIA CUDA
  Toolkit and the NVIDIA CUDA Deep Neural Network library (or a
  modified version of those libraries), containing parts covered by the
  terms of the respective license agreement, the licensors of this
  Program grant you additional permission to convey the resulting work.
*/

#include "cuda_common.h"
#include "neural/shared/activation.h"

// Allow building on an old architecture.
#if __CUDA_ARCH__ < 530
#define SKIP_FP16_BITS 1
#endif
#include "winograd_helper.inc"

namespace lczero {
namespace cudnn_backend {

/////////////////////////////////////////////////////////////////////////////
//          fp16-specific kernels used by certain layers                   //
/////////////////////////////////////////////////////////////////////////////

// SE layer implementation using single fused kernel.

// N blocks.
// C threads per block.
// 'HWC' input data processed by thread block.
// Each thread processes 8x8 elements.
// K is the no. of outputs of first fully connected layer (same as no. of inputs
// for second fully connected layer).
// The kernel assumes K <= C.

template <int C, int K>
__global__ void SE_Layer_NHWC(half* output, const half* skip, const half* input,
                              const half* w1, const half* b1, const half* w2,
                              const half* b2, const half* bPrev,
                              ActivationFunction activation) {
#if __CUDA_ARCH__ >= 530
  const int elementsPerThread = 64;  // 8x8 board
  const int se_K = K;

  int n = blockIdx.x;
  int c = threadIdx.x;

  __shared__ half sharedData[C];

  half2 localData[elementsPerThread];

  half S = 0;

  half bias = 0;
  if (bPrev) bias = bPrev[c];

// 1. Global avg (1 avg per thread).
#pragma unroll
  for (int i = 0; i < elementsPerThread; i++) {
    int localIndex = i * C + c;
    int inputIndex = n * C * elementsPerThread + localIndex;
    localData[i].x = input[inputIndex] + bias;
    localData[i].y = skip[inputIndex];
    S += localData[i].x;
  }

  half avg = S / (half)elementsPerThread;
  sharedData[c] = avg;

  __syncthreads();

  // 2. First fully connected layer.
  if (c < K) {
    S = 0;

#pragma unroll
    for (int i = 0; i < C; i++) {
      S += sharedData[i] * readw1(i, c);
    }

    S += b1[c];

    S = activate(S, activation);

    sharedData[c] = S;
  }
  __syncthreads();

  // 3. Second fully connected layer.
  S = 0;
  half B = 0;
#pragma unroll
  for (int i = 0; i < K; i++) {
    half val = sharedData[i];
    S += val * readw2(i, c);
    B += val * readw2(i, c + C);
  }
  S += b2[c];
  B += b2[c + C];

  // Sigmoid (only on the scale part).
  S = (half)(1.0f / (1.0f + exp(-(float)(S))));

// 4. Scale, and add skip connection, perform relu, and write to output.
#pragma unroll
  for (int i = 0; i < elementsPerThread; i++) {
    int localIndex = i * C + c;
    int inputIndex = n * C * elementsPerThread + localIndex;
    half val = localData[i].y + localData[i].x * S + B;

    // Relu activation function.
    val = (half)activate((float)val, activation);

    output[inputIndex] = val;
  }
#endif
}

bool Se_Fp16_NHWC(int N, int C, int numFc1Out, half* output, const half* skip,
                  const half* input, const half* w1, const half* b1,
                  const half* w2, const half* b2, const half* bPrev,
                  ActivationFunction activation) {
  // TODO: Think of more elegant way to avoid this hardcoding :-/
  if (numFc1Out == 16) {
    if (C == 64) {
      SE_Layer_NHWC<64, 16>
          <<<N, C>>>(output, skip, input, w1, b1, w2, b2, bPrev, activation);
    } else {
      // TODO: support other channel counts.
      throw Exception("channel count unsupported by SE layer");
    }
  } else if (numFc1Out == 32) {
    if (C == 64) {
      SE_Layer_NHWC<64, 32>
          <<<N, C>>>(output, skip, input, w1, b1, w2, b2, bPrev, activation);
    } else if (C == 128) {
      SE_Layer_NHWC<128, 32>
          <<<N, C>>>(output, skip, input, w1, b1, w2, b2, bPrev, activation);
    } else if (C == 192) {
      SE_Layer_NHWC<192, 32>
          <<<N, C>>>(output, skip, input, w1, b1, w2, b2, bPrev, activation);
    } else if (C == 256) {
      SE_Layer_NHWC<256, 32>
          <<<N, C>>>(output, skip, input, w1, b1, w2, b2, bPrev, activation);
    } else if (C == 320) {
      SE_Layer_NHWC<320, 32>
          <<<N, C>>>(output, skip, input, w1, b1, w2, b2, bPrev, activation);
    } else if (C == 352) {
      SE_Layer_NHWC<352, 32>
          <<<N, C>>>(output, skip, input, w1, b1, w2, b2, bPrev, activation);
    } else if (C == 384) {
      SE_Layer_NHWC<384, 32>
          <<<N, C>>>(output, skip, input, w1, b1, w2, b2, bPrev, activation);
    } else {
      // TODO: support other channel counts.
      return false;
    }
  } else if (numFc1Out == 64) {
    if (C == 64) {
      SE_Layer_NHWC<64, 64>
          <<<N, C>>>(output, skip, input, w1, b1, w2, b2, bPrev, activation);
    } else if (C == 128) {
      SE_Layer_NHWC<128, 64>
          <<<N, C>>>(output, skip, input, w1, b1, w2, b2, bPrev, activation);
    } else if (C == 192) {
      SE_Layer_NHWC<192, 64>
          <<<N, C>>>(output, skip, input, w1, b1, w2, b2, bPrev, activation);
    } else if (C == 256) {
      SE_Layer_NHWC<256, 64>
          <<<N, C>>>(output, skip, input, w1, b1, w2, b2, bPrev, activation);
    } else if (C == 320) {
      SE_Layer_NHWC<320, 64>
          <<<N, C>>>(output, skip, input, w1, b1, w2, b2, bPrev, activation);
    } else if (C == 384) {
      SE_Layer_NHWC<384, 64>
          <<<N, C>>>(output, skip, input, w1, b1, w2, b2, bPrev, activation);
    } else {
      // TODO: support other channel counts.
      return false;
    }
  } else {
    // TODO: support other sizes.
    return false;
  }
  ReportCUDAErrors(hipGetLastError());
  return true;
}

// Get board for this thread from shared memory.
// We are just using shared memory to store local thread data in this kernel to
// help reduce some register pressure and spills to local memory.
#define BOARD(y, x) shboard[(y)*8 + (x)]

// input is in transformed space (HWNC layout) --- output of GEMM
// output is also in transformed space (HWNC layout) --- input to GEMM (for
// next layer)
// 'C' threads per block
// 'N' blocks
// Every thread generates an entire board/plane (8x8 elements).
template <ActivationFunction activation, bool use_bias, bool use_skip>
__global__ __launch_bounds__(
    kMaxResBlockFusingSeKFp16Ampere,
    1) void OutputInputTransformKernel_fp16_shmem_board(int N, int C, int se_K,
                                                        half* output,
                                                        const half* input,
                                                        half* skip,
                                                        const half* bias,
                                                        const half* w1,
                                                        const half* b1,
                                                        const half* w2,
                                                        const half* b2) {
#if __CUDA_ARCH__ >= 530
  int k = threadIdx.x;
  int n = blockIdx.x;

  extern __shared__ half _sboard[];
  half* shboard = &_sboard[k * 72];  // 72 instead of 64 to reduce shared
                                     // memory bank conflicts.
  half b = bias[k];

#pragma unroll
  for (int hStart = 0; hStart < 8; hStart += 4)
#pragma unroll
    for (int wStart = 0; wStart < 8; wStart += 4) {
      //  i) read to per thread registers (for doing output transform)
      int shln = n * 4 + (hStart / 4) * 2 + (wStart / 4);
      half outElTransformed[6][6];
#pragma unroll
      for (int y = 0; y < 6; y++)
#pragma unroll
        for (int x = 0; x < 6; x++)
          outElTransformed[y][x] = input[TEMP_INDEX_HWNC(y, x, shln, k)];

      // ii) transform it
      half outEl[4][4];
      OutputTransform4x4(&outEl[0][0], &outElTransformed[0][0]);

#pragma unroll
      for (int y = 0; y < 4; y++)
        copyAs<uint2>(&BOARD(hStart + y, wStart), &outEl[y][0]);
    }

  // Add bias, and compute the average for SE.
  float S = 0;
  float B = 0;

#pragma unroll
  for (int y = 0; y < 8; y++) {
    half boardRow[8];
    copyAs<uint4>(&boardRow, &BOARD(y, 0));
#pragma unroll
    for (int x = 0; x < 8; x++) {
      if (use_bias) boardRow[x] += b;
      S += (float)boardRow[x];
    }
    if (use_bias) copyAs<uint4>(&BOARD(y, 0), &boardRow);
  }

  __shared__ float shared_data[kMaxResBlockFusingSeKFp16Ampere];
  float avg = S / 64;
  shared_data[k] = avg;

  int lane = k & 0x1F;
  int warp = k >> 5;
  __syncthreads();

  // First fully-connected layer for SE

  // As se_K << C, we want to loop over se_K instead of C
  // even if it means taking the sum across threads

  __shared__ float shared_sums[kMaxResBlockFusingSeKFp16Ampere / 32]
                              [kMaxResBlockFusingSeK];  // per-warp sums

  for (int i = 0; i < se_K; i++) {
    float val = shared_data[k] * float(readw1(k, i));
    val = warpReduce(val);
    if (lane == 0) shared_sums[warp][i] = val;
  }
  __syncthreads();
  if (k < se_K) {
    S = 0;
    for (int i = 0; i < C / 32; i++) S += shared_sums[i][k];

    S += (float)b1[k];
    S = activate(S, activation);
    shared_data[k] = S;
  }

  __syncthreads();

  // Second fully-connected layer for SE
  S = 0;
  for (int i = 0; i < se_K; i++) {
    float val = shared_data[i];
    S += val * float(readw2(i, k));
    B += val * float(readw2(i, k + C));
  }
  S += (float)b2[k];
  B += (float)b2[k + C];

  // Sigmoid (only on the scale part).
  S = 1.0f / (1.0f + exp(-S));

  // Scale/bias, add skip connection, perform activation, and write to output.
  for (int h = 0; h < 8; h++) {
    half boardRow[8];
    copyAs<uint4>(&boardRow[0], &BOARD(h, 0));

#pragma unroll
    for (int w = 0; w < 8; w++) {
      boardRow[w] = (half)(float(boardRow[w]) * S + B);
    }

    // residual add
    if (use_skip) {
      half skipInp[8];
      copyAs<uint4>(&skipInp[0], &skip[INDEX_NHCW(n, k, h, 0)]);
#pragma unroll
      for (int w = 0; w < 8; w++) boardRow[w] += skipInp[w];
    }

    if (activation != ACTIVATION_NONE) {
#pragma unroll
      for (int w = 0; w < 8; w++)
        boardRow[w] = (half)activate((float)boardRow[w], activation);
    }

    // write un-transformed output to 'skip' if required
    if (use_skip) {
      copyAs<uint4>(&skip[INDEX_NHCW(n, k, h, 0)], &boardRow[0]);
    }

    copyAs<uint4>(&BOARD(h, 0), &boardRow);
  }

  // Perform input transform.

  int c = k;
  // top-left
  {
    half inEl[6][6] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                       0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

#pragma unroll
    for (int i = 0; i < 5; i++)
#pragma unroll
      for (int j = 0; j < 5; j++) inEl[i + 1][j + 1] = BOARD(i, j);

    InputTransform4x4(&inEl[0][0], &inEl[0][0]);

#pragma unroll
    for (int y = 0; y < 6; y++)
#pragma unroll
      for (int x = 0; x < 6; x++)
        output[TEMP_INDEX_HWNC(y, x, n * 4 + 0, c)] = inEl[y][x];
  }

  // top-right
  {
    half inEl[6][6] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                       0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

#pragma unroll
    for (int i = 0; i < 5; i++)
#pragma unroll
      for (int j = 0; j < 5; j++) inEl[i + 1][j] = BOARD(i, j + 3);

    InputTransform4x4(&inEl[0][0], &inEl[0][0]);

#pragma unroll
    for (int y = 0; y < 6; y++)
#pragma unroll
      for (int x = 0; x < 6; x++)
        output[TEMP_INDEX_HWNC(y, x, n * 4 + 1, c)] = inEl[y][x];
  }

  // bottom-left
  {
    half inEl[6][6] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                       0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

#pragma unroll
    for (int i = 0; i < 5; i++)
#pragma unroll
      for (int j = 0; j < 5; j++) inEl[i][j + 1] = BOARD(i + 3, j);

    InputTransform4x4(&inEl[0][0], &inEl[0][0]);

#pragma unroll
    for (int y = 0; y < 6; y++)
#pragma unroll
      for (int x = 0; x < 6; x++)
        output[TEMP_INDEX_HWNC(y, x, n * 4 + 2, c)] = inEl[y][x];
  }

  // bottom-right
  {
    half inEl[6][6] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                       0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

#pragma unroll
    for (int i = 0; i < 5; i++)
#pragma unroll
      for (int j = 0; j < 5; j++) inEl[i][j] = BOARD(i + 3, j + 3);

    InputTransform4x4(&inEl[0][0], &inEl[0][0]);

#pragma unroll
    for (int y = 0; y < 6; y++)
#pragma unroll
      for (int x = 0; x < 6; x++)
        output[TEMP_INDEX_HWNC(y, x, n * 4 + 3, c)] = inEl[y][x];
  }
#endif
}

template <typename T = half, bool use_se, ActivationFunction activation,
          bool use_bias, bool use_skip>
void OutputInputTransform(int N, int C, int se_K, T* output, const T* input,
                          const T* skip, const T* bias, const T* w1,
                          const T* b1, const T* w2, const T* b2,
                          hipStream_t stream) {
  // Each thread processes entire chess board.
  if (use_se == false) {
    dim3 grid_dim(DivUp(C, kOpInpTransformBlockSize), N, 1);
    OutputTransform_relu_InputTransform_kernel<half, activation, use_bias,
                                               use_skip>
        <<<grid_dim, kOpInpTransformBlockSize, 0, stream>>>(N, C, output, input,
                                                            (half*)skip, bias);
  } else if (C > kMaxResBlockFusingChannels) {
    // Use special kernel with reduced register pressure - only works on Ampere,
    // and only for fp16.
    if (C <= kMaxResBlockFusingSeKFp16Ampere) {
      hipFuncSetAttribute(reinterpret_cast<const void*>(
          OutputInputTransformKernel_fp16_shmem_board<activation), use_bias,
                                                      use_skip>,
          hipFuncAttributeMaxDynamicSharedMemorySize, 72 * C * sizeof(half));
      OutputInputTransformKernel_fp16_shmem_board<activation, use_bias,
                                                  use_skip>
          <<<N, C, 72 * C * sizeof(half), stream>>>(
              N, C, se_K, (half*)output, (const half*)input, (half*)skip,
              (half*)bias, (half*)w1, (half*)b1, (half*)w2, (half*)b2);
    } else {
      throw Exception(
          "res block fusing opt not supported for the given data type and no "
          "of filters\n");
    }
  } else {
    OutputTransform_SE_relu_InputTransform_kernel<half, activation, use_bias,
                                                  use_skip>
        <<<N, C, 0, stream>>>(N, C, se_K, output, input, (half*)skip, bias, w1,
                              b1, w2, b2);
  }
  ReportCUDAErrors(hipGetLastError());
}

template void FilterTransform<half>(int N, int C, half* transformedFilter,
                                    const half* filter);

template void InputTransform<half, true>(int N, int C, half* transformed_input,
                                         const half* input,
                                         hipStream_t stream);
template void InputTransform<half, false>(int N, int C, half* transformed_input,
                                          const half* input,
                                          hipStream_t stream);

template void OutputTransform<half, true, ACTIVATION_RELU, true, true, false,
                              false>(int N, int C, int se_K, half* output,
                                     const half* input, const half* skip,
                                     const half* bias, const half* w1,
                                     const half* b1, const half* w2,
                                     const half* b2, hipStream_t stream);

template void OutputTransform<half, false, ACTIVATION_RELU, true, true, false,
                              false>(int N, int C, int se_K, half* output,
                                     const half* input, const half* skip,
                                     const half* bias, const half* w1,
                                     const half* b1, const half* w2,
                                     const half* b2, hipStream_t stream);

template void OutputTransform<half, true, ACTIVATION_RELU, true, true, true,
                              false>(int N, int C, int se_K, half* output,
                                     const half* input, const half* skip,
                                     const half* bias, const half* w1,
                                     const half* b1, const half* w2,
                                     const half* b2, hipStream_t stream);

template void OutputTransform<half, false, ACTIVATION_RELU, true, true, true,
                              false>(int N, int C, int se_K, half* output,
                                     const half* input, const half* skip,
                                     const half* bias, const half* w1,
                                     const half* b1, const half* w2,
                                     const half* b2, hipStream_t stream);

template void OutputTransform<half, false, ACTIVATION_RELU, true, false, false,
                              false>(int N, int C, int se_K, half* output,
                                     const half* input, const half* skip,
                                     const half* bias, const half* w1,
                                     const half* b1, const half* w2,
                                     const half* b2, hipStream_t stream);

template void OutputTransform<half, false, ACTIVATION_RELU, true, false, false,
                              true>(int N, int C, int se_K, half* output,
                                    const half* input, const half* skip,
                                    const half* bias, const half* w1,
                                    const half* b1, const half* w2,
                                    const half* b2, hipStream_t stream);

template void OutputTransform<half, true, ACTIVATION_RELU, true, true, true,
                              true>(int N, int C, int se_K, half* output,
                                    const half* input, const half* skip,
                                    const half* bias, const half* w1,
                                    const half* b1, const half* w2,
                                    const half* b2, hipStream_t stream);

template void OutputTransform<half, true, ACTIVATION_MISH, true, true, false,
                              false>(int N, int C, int se_K, half* output,
                                     const half* input, const half* skip,
                                     const half* bias, const half* w1,
                                     const half* b1, const half* w2,
                                     const half* b2, hipStream_t stream);

template void OutputTransform<half, false, ACTIVATION_MISH, true, true, false,
                              false>(int N, int C, int se_K, half* output,
                                     const half* input, const half* skip,
                                     const half* bias, const half* w1,
                                     const half* b1, const half* w2,
                                     const half* b2, hipStream_t stream);

template void OutputTransform<half, true, ACTIVATION_MISH, true, true, true,
                              false>(int N, int C, int se_K, half* output,
                                     const half* input, const half* skip,
                                     const half* bias, const half* w1,
                                     const half* b1, const half* w2,
                                     const half* b2, hipStream_t stream);

template void OutputTransform<half, false, ACTIVATION_MISH, true, true, true,
                              false>(int N, int C, int se_K, half* output,
                                     const half* input, const half* skip,
                                     const half* bias, const half* w1,
                                     const half* b1, const half* w2,
                                     const half* b2, hipStream_t stream);

template void OutputTransform<half, false, ACTIVATION_MISH, true, false, false,
                              false>(int N, int C, int se_K, half* output,
                                     const half* input, const half* skip,
                                     const half* bias, const half* w1,
                                     const half* b1, const half* w2,
                                     const half* b2, hipStream_t stream);

template void OutputTransform<half, false, ACTIVATION_MISH, true, false, false,
                              true>(int N, int C, int se_K, half* output,
                                    const half* input, const half* skip,
                                    const half* bias, const half* w1,
                                    const half* b1, const half* w2,
                                    const half* b2, hipStream_t stream);

template void OutputTransform<half, true, ACTIVATION_MISH, true, true, true,
                              true>(int N, int C, int se_K, half* output,
                                    const half* input, const half* skip,
                                    const half* bias, const half* w1,
                                    const half* b1, const half* w2,
                                    const half* b2, hipStream_t stream);

template void OutputTransform<half, false, ACTIVATION_NONE, true, false, false,
                              false>(int N, int C, int se_K, half* output,
                                     const half* input, const half* skip,
                                     const half* bias, const half* w1,
                                     const half* b1, const half* w2,
                                     const half* b2, hipStream_t stream);

template void OutputInputTransform<half, true, ACTIVATION_RELU, true, true>(
    int N, int C, int se_K, half* output, const half* input, const half* skip,
    const half* bias, const half* w1, const half* b1, const half* w2,
    const half* b2, hipStream_t stream);

template void OutputInputTransform<half, false, ACTIVATION_RELU, true, true>(
    int N, int C, int se_K, half* output, const half* input, const half* skip,
    const half* bias, const half* w1, const half* b1, const half* w2,
    const half* b2, hipStream_t stream);

template void OutputInputTransform<half, false, ACTIVATION_RELU, true, false>(
    int N, int C, int se_K, half* output, const half* input, const half* skip,
    const half* bias, const half* w1, const half* b1, const half* w2,
    const half* b2, hipStream_t stream);

template void OutputInputTransform<half, true, ACTIVATION_MISH, true, true>(
    int N, int C, int se_K, half* output, const half* input, const half* skip,
    const half* bias, const half* w1, const half* b1, const half* w2,
    const half* b2, hipStream_t stream);

template void OutputInputTransform<half, false, ACTIVATION_MISH, true, true>(
    int N, int C, int se_K, half* output, const half* input, const half* skip,
    const half* bias, const half* w1, const half* b1, const half* w2,
    const half* b2, hipStream_t stream);

template void OutputInputTransform<half, false, ACTIVATION_MISH, true, false>(
    int N, int C, int se_K, half* output, const half* input, const half* skip,
    const half* bias, const half* w1, const half* b1, const half* w2,
    const half* b2, hipStream_t stream);

}  // namespace cudnn_backend
}  // namespace lczero
